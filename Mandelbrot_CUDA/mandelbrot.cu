#include "hip/hip_runtime.h"

// Mandelbrot
// CUDA OpenGL Interoperability
//
// This program requires nVidia GPU and CUDA Toollkit

#include "framework.h"
#include "matrix.h"
#include "shader.h"
#include "square1.h"
#include "square2.h"
#include "stack.h"

const UINT_PTR IDM_OPEN = 111;
const UINT_PTR IDM_SAVE_AS = 112;
const UINT_PTR IDM_EXIT = 113;

const int MAX_ITERATION_COUNT = 32768;

const int PARAM_1_COUNT = 4;
const int PARAM_2_COUNT = 3;
const int PARAM_3_COUNT = 3 * MAX_ITERATION_COUNT;

const unsigned long long PARAM_1_SIZE = PARAM_1_COUNT * sizeof(double);
const unsigned long long PARAM_2_SIZE = PARAM_2_COUNT * sizeof(int);
const unsigned long long PARAM_3_SIZE = PARAM_3_COUNT * sizeof(unsigned char);

const UINT_PTR IDM_FRACTAL = 121;
const UINT_PTR IDM_GO_BACK = 122;

const int MAX_LOADSTRING = 100;

const int FRACTAL_WIDTH = 1280;
const int FRACTAL_HEIGHT = 720;

// Global Variables:
HINSTANCE hInst;                                // current instance
CHAR szTitle[MAX_LOADSTRING];                  // The title bar text
CHAR szWindowClass[MAX_LOADSTRING];            // main window class name
HMENU hMenu;

CMatrix matrix;
CShader shader1, shader2;
CSquare1 square1;
CSquare2 square2;
CStack stack;
GLuint textures;
double* Param1;
int* Param2;
unsigned char* Param3;
int height, px, py, px1, py1, px2, py2;
bool is_dragging, show_selection;

// Forward declarations of functions included in this code module:
LRESULT CALLBACK    WndProc(HWND hWnd, UINT message, WPARAM wParam, LPARAM lParam);

void OnAfterWindowDisplayed(HWND hWnd, WPARAM wParam, LPARAM lParam);

HMENU CreateAppMenu(HWND hWnd);
void DestroyAppMenu(HMENU hMenu);

void GetUniqueName(wchar_t* filename, DWORD size);
void CreateColorGradient(unsigned char* gradient, int count);
__global__ void DoFractal(double* param1, int* param2, unsigned char* param3, hipSurfaceObject_t object);
void DoParallelComputing(double* Param1, size_t size1, int* Param2, size_t size2, unsigned char* Param3, size_t size3);

void OnLButtonDown(HWND hWnd, WORD Key, int x, int y);
void OnLButtonUp(HWND hWnd, WORD Key, int x, int y);
void OnMouseMove(HWND hWnd, WORD Key, int x, int y);
void OnSize(HWND hWnd, int width, int height);

void OnPaint(HDC hDC);
void OnCreate(HWND hWnd, HDC* hDC);
void OnDestroy(HWND hWnd, HDC hDC);

void OnFileOpen(HWND hWnd);
void OnFileSaveAs(HWND hWnd);
void OnFileExit(HWND hWnd);

void OnViewFractal(HWND hWnd);
void OnViewGoBack(HWND hWnd);

int main()
{
    HINSTANCE hInstance;
    WNDCLASSEXA wcex;
    HWND hWnd;
    MSG msg;
    int X, Y, nWidth, nHeight, Cx, Cy;

    strcpy_s(szTitle, "Mandelbrot");
    strcpy_s(szWindowClass, "MandelbrotClass");

    hInst = hInstance = GetModuleHandle(NULL);

    wcex.cbSize = sizeof(WNDCLASSEX);
    wcex.style = CS_HREDRAW | CS_VREDRAW;
    wcex.lpfnWndProc = WndProc;
    wcex.cbClsExtra = 0;
    wcex.cbWndExtra = 0;
    wcex.hInstance = hInstance;
    wcex.hIcon = LoadIcon(NULL, IDI_APPLICATION);;
    wcex.hCursor = LoadCursor(NULL, IDC_ARROW);
    wcex.hbrBackground = NULL;
    wcex.lpszMenuName = NULL;
    wcex.lpszClassName = szWindowClass;
    wcex.hIconSm = LoadIcon(NULL, IDI_APPLICATION);;

    if (!RegisterClassExA(&wcex)) return 0;

    X = 200;
    Y = 100;

    Cx = FRACTAL_WIDTH;
    Cy = FRACTAL_HEIGHT;

    nWidth = Cx + 16;
    nHeight = Cy + 59;

    hWnd = CreateWindowExA(NULL,
        szWindowClass,
        szTitle,
        WS_OVERLAPPEDWINDOW,
        X, Y,
        nWidth, nHeight,
        NULL,
        NULL,
        hInstance,
        NULL);

    if (!hWnd) return 0;

    ShowWindow(hWnd, SW_SHOW);
    UpdateWindow(hWnd);

    while (GetMessage(&msg, NULL, 0, 0))
    {
        TranslateMessage(&msg);
        DispatchMessage(&msg);
    }

    return (int)msg.wParam;
}

// Processes messages for the main window.
LRESULT CALLBACK WndProc(HWND hWnd, UINT message, WPARAM wParam, LPARAM lParam)
{
    static HDC hDC;

    switch (message)
    {
    case WM_AFTERWINDOWDISPLAYED: OnAfterWindowDisplayed(hWnd, wParam, lParam);  break;

    case WM_COMMAND:
        switch (LOWORD(wParam))
        {
        case IDM_OPEN:	            OnFileOpen(hWnd);		                    break;
        case IDM_SAVE_AS:	        OnFileSaveAs(hWnd);		                    break;
        case IDM_EXIT:	            OnFileExit(hWnd);		                    break;
        case IDM_FRACTAL:           OnViewFractal(hWnd);        break;
        case IDM_GO_BACK:           OnViewGoBack(hWnd);         break;
        default:
            return DefWindowProc(hWnd, message, wParam, lParam);
        }
        break;
    case WM_LBUTTONDOWN:    OnLButtonDown(hWnd, WORD(wParam), LOWORD(lParam), HIWORD(lParam));  break;
    case WM_LBUTTONUP:      OnLButtonUp(hWnd, WORD(wParam), LOWORD(lParam), HIWORD(lParam));    break;
    case WM_MOUSEMOVE:      OnMouseMove(hWnd, WORD(wParam), LOWORD(lParam), HIWORD(lParam));    break;
    case WM_SIZE:           OnSize(hWnd, LOWORD(lParam), HIWORD(lParam));                    break;
    case WM_PAINT:          OnPaint(hDC);                                                       break;
    case WM_CREATE:         OnCreate(hWnd, &hDC);                                               break;
    case WM_DESTROY:        OnDestroy(hWnd, hDC);                                               break;
    default:
        return DefWindowProc(hWnd, message, wParam, lParam);
    }
    return 0;
}

void OnAfterWindowDisplayed(HWND hWnd, WPARAM wParam, LPARAM lParam)
{
    DoParallelComputing(Param1, PARAM_1_SIZE, Param2, PARAM_2_SIZE, Param3, PARAM_3_SIZE);
}

// Adding Lines and Graphs to a Menu
// https://learn.microsoft.com/en-us/windows/win32/menurc/using-menus
HMENU CreateAppMenu(HWND hWnd)
{
    HMENU hMenu, hFile, hView;

    hMenu = CreateMenu();
    hFile = CreatePopupMenu();
    hView = CreatePopupMenu();

    AppendMenu(hMenu, MF_STRING | MF_POPUP, (UINT_PTR)hFile, "File");
    AppendMenu(hFile, MF_STRING, IDM_OPEN, "Open...");
    AppendMenu(hFile, MF_STRING, IDM_SAVE_AS, "Save As...");
    AppendMenu(hFile, MF_SEPARATOR, 0, NULL);
    AppendMenu(hFile, MF_STRING, IDM_EXIT, "Exit");

    AppendMenu(hMenu, MF_STRING | MF_POPUP, (UINT_PTR)hView, "View");
    AppendMenu(hView, MF_STRING, IDM_FRACTAL, "Fractal");
    AppendMenu(hView, MF_STRING, IDM_GO_BACK, "Go Back");

    SetMenu(hWnd, hMenu);

    return hMenu;
}

void DestroyAppMenu(HMENU hMenu)
{
    DestroyMenu(hMenu);
}

void GetUniqueName(wchar_t* filename, DWORD size)
{
    time_t ltime;
    struct tm a;

    time(&ltime);
    _localtime64_s(&a, &ltime);

    swprintf_s(filename, size, L"%d%02d%02d%02d%02d%02d.mdl", a.tm_year + 1900, a.tm_mon + 1, a.tm_mday, a.tm_hour, a.tm_min, a.tm_sec);
}

// gradient is a series of rgb value
void CreateColorGradient(unsigned char* gradient, int count)
{
    int i, k, quo, c1, c2, c3;

    k = 0;

    for (i = 0; i < count; i++) {

        quo = i / 32;

        c3 = i % 32;
        c2 = quo % 32;
        c1 = quo / 32;

        gradient[k++] = (unsigned char)(255.0 * ((double)c1 / 31.0));
        gradient[k++] = (unsigned char)(255.0 * ((double)c2 / 31.0));
        gradient[k++] = (unsigned char)(255.0 * ((double)c3 / 31.0));
    }
}

//  param1[0] - cx
//  param1[1] - cy
//  param1[2] - ox
//  param1[3] - oy
//
//  param2[0] - FRACTAL_WIDTH
//  param2[1] - FRACTAL_HEIGHT
//  param2[2] - MAX_ITERATION_COUNT
//
//  param3[0] - color gradient

__global__ void DoFractal(double* param1, int* param2, unsigned char* param3, hipSurfaceObject_t object)
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    int i, j, k;
    double ax, ay, bx, by, rx, ry;
    unsigned char r, g, b;

    if (y < param2[1]) {

        bx = ((double)x / (double)(param2[0] - 1)) * param1[0] + param1[2];
        by = ((double)y / (double)(param2[1] - 1)) * param1[1] + param1[3];

        ax = 0.0;
        ay = 0.0;

        j = 0;

        for (i = 0; i < param2[2]; i++) {

            rx = ax * ax - ay * ay + bx;
            ry = 2.0 * ax * ay + by;

            if ((rx * rx + ry * ry) > 4.0) {
                j = i;
                break;
            }

            ax = rx;
            ay = ry;
        }

        k = 3 * j;
        
        r = param3[k];
        g = param3[k + 1];
        b = param3[k + 2];

        surf2Dwrite(make_uchar4(r, g, b, 0), object, x * sizeof(uchar4), y);
    }
}

void DoParallelComputing(double* Param1, size_t size1, int* Param2, size_t size2, unsigned char* Param3, size_t size3)
{
    LARGE_INTEGER freq, t1, t2;
    LONGLONG tmi, quo, ms, ss, mm, hh;
    hipGraphicsResource* resource;
    hipError_t result;
    hipArray_t pointer;
    hipResourceDesc descriptor;
    hipSurfaceObject_t object;
    double* param1;
    int* param2;
    unsigned char* param3;

    QueryPerformanceFrequency(&freq);
    QueryPerformanceCounter(&t1);

    // 1280 / 32 = 40   -> 40
    //  720 / 32 = 22.5 -> 23 -> check for out of index

    dim3 grid_size(40, 23);     // the number of thread blocks in the grid
    dim3 block_size(32, 32);    // the number of threads in a thread block

    result = hipGraphicsGLRegisterImage(&resource, textures, GL_TEXTURE_2D, 0);

    if (result == hipSuccess) {

        result = hipGraphicsMapResources(1, &resource, 0);

        if (result == hipSuccess) {

            result = hipGraphicsSubResourceGetMappedArray(&pointer, resource, 0, 0);

            if (result == hipSuccess) {

                ZeroMemory(&descriptor, sizeof(descriptor));

                descriptor.resType = hipResourceTypeArray;
                descriptor.res.array.array = pointer;

                result = hipCreateSurfaceObject(&object, &descriptor);

                if (result == hipSuccess) {

                    hipMalloc(&param1, size1);
                    hipMalloc(&param2, size2);
                    hipMalloc(&param3, size3);

                    hipMemcpy(param1, Param1, size1, hipMemcpyHostToDevice);
                    hipMemcpy(param2, Param2, size2, hipMemcpyHostToDevice);
                    hipMemcpy(param3, Param3, size3, hipMemcpyHostToDevice);

                    DoFractal << < grid_size, block_size >> > (param1, param2, param3, object);

                    hipFree(param1);
                    hipFree(param2);
                    hipFree(param3);

                    hipDestroySurfaceObject(object);

                    result = hipDeviceSynchronize();

                    if (result != hipSuccess) {
                        OutputDebugStringA("hipDeviceSynchronize error - ");
                        OutputDebugStringA(hipGetErrorName(hipGetLastError()));
                        OutputDebugStringA("\n");
                    }
                }
                else {
                    OutputDebugStringA("hipCreateSurfaceObject error - ");
                    OutputDebugStringA(hipGetErrorName(hipGetLastError()));
                    OutputDebugStringA("\n");
                }
            }
            else {
                OutputDebugStringA("hipGraphicsSubResourceGetMappedArray error - ");
                OutputDebugStringA(hipGetErrorName(hipGetLastError()));
                OutputDebugStringA("\n");
            }

            hipGraphicsUnmapResources(1, &resource, 0);
        }
        else {
            OutputDebugStringA("hipGraphicsMapResources error - ");
            OutputDebugStringA(hipGetErrorName(hipGetLastError()));
            OutputDebugStringA("\n");
        }
    }
    else {
        OutputDebugStringA("hipGraphicsGLRegisterImage error - ");
        OutputDebugStringA(hipGetErrorName(hipGetLastError()));
        OutputDebugStringA("\n");
    }

    QueryPerformanceCounter(&t2);
    tmi = ((t2.QuadPart - t1.QuadPart) * 1000LL) / freq.QuadPart;

    quo = tmi / 1000;
    ms = tmi % 1000;

    tmi = quo;

    quo = tmi / 60;
    ss = tmi % 60;

    tmi = quo;

    hh = tmi / 60;
    mm = tmi % 60;

    printf("%lld:%02lld:%02lld:%03lld\n", hh, mm, ss, ms);
}


void OnLButtonDown(HWND hWnd, WORD Key, int x, int y)
{
    EnableMenuItem(hMenu, IDM_FRACTAL, MF_BYCOMMAND | MF_DISABLED);

    is_dragging = true;
    show_selection = false;

    px = x;
    py = y;
}

void OnLButtonUp(HWND hWnd, WORD Key, int x, int y)
{
    is_dragging = false;
}

void OnMouseMove(HWND hWnd, WORD Key, int x, int y)
{
    if (is_dragging) {

        // arrange coordinate in ascending order
        if (px > x) {
            px1 = x;
            px2 = px;
        }
        else {
            px1 = px;
            px2 = x;
        }

        if (py > y) {
            py2 = height - y;
            py1 = height - py;
        }
        else {
            py2 = height - py;
            py1 = height - y;
        }

        // enable this menu if there is a selection
        if ((px2 - px1) > 2 && (py2 - py1) > 2) {

            EnableMenuItem(hMenu, IDM_FRACTAL, MF_BYCOMMAND | MF_ENABLED);

            show_selection = true;
            square2.Update((float)px1, (float)py1, (float)(px2), (float)py2);
        }
    }
}

void OnSize(HWND hWnd, int width, int height)
{
    printf("%10d%10d\n", width, height);

    ::height = height;

    float left, right, bottom, top, znear, zfar;

    left = 0.0f;
    right = (float)width;

    bottom = 0.0f;
    top = (float)height;

    znear = 0.0f;
    zfar = 1.0f;

    matrix.Orthographic(left, right, bottom, top, znear, zfar);

    glViewport(0, 0, width, height);
}

void OnPaint(HDC hDC)
{
    glClear(GL_COLOR_BUFFER_BIT);

    shader1.Use();
    square1.Draw(matrix);

    if (show_selection) {
        shader2.Use();
        square2.Draw(matrix);
    }

    SwapBuffers(hDC);
}

void OnCreate(HWND hWnd, HDC* hDC)
{
    PIXELFORMATDESCRIPTOR pfd;
    int format;
    HGLRC hglrc;

    pfd.nSize = sizeof(PIXELFORMATDESCRIPTOR);  // size of structured data
    pfd.nVersion = 1;                           // version number
    pfd.dwFlags = PFD_DRAW_TO_WINDOW |          // support window
        PFD_SUPPORT_OPENGL |                    // support OpenGL
        PFD_DOUBLEBUFFER |                      // double buffered
        PFD_GENERIC_ACCELERATED;                // support device driver that accelerates the generic implementation
    pfd.iPixelType = PFD_TYPE_RGBA;             // RGBA pixels
    pfd.cColorBits = 24;                        //  24-bit color
    pfd.cRedBits = 0;                           // color bits ignored
    pfd.cGreenBits = 0;
    pfd.cBlueBits = 0;
    pfd.cAlphaBits = 0;
    pfd.cRedShift = 0;                          // shift bit ignored
    pfd.cGreenShift = 0;
    pfd.cBlueShift = 0;
    pfd.cAlphaShift = 0;
    pfd.cAccumBits = 0;                         // no accumulation buffer
    pfd.cAccumRedBits = 0;
    pfd.cAccumGreenBits = 0;
    pfd.cAccumBlueBits = 0;
    pfd.cAccumAlphaBits = 0;
    pfd.cDepthBits = 32;                        // 32-bit z-buffer
    pfd.cStencilBits = 0;                       // no stencil buffer
    pfd.cAuxBuffers = 0;                        // no auxiliary buffer
    pfd.iLayerType = PFD_MAIN_PLANE;            // main layer
    pfd.bReserved = 0;
    pfd.dwLayerMask = 0;
    pfd.dwVisibleMask = 0;
    pfd.dwDamageMask = 0;

    *hDC = GetDC(hWnd);                         // get the device context for our window
    format = ChoosePixelFormat(*hDC, &pfd); // get the best available match of pixel format for the device context
    SetPixelFormat(*hDC, format, &pfd);     // make that the pixel format of the device context
    hglrc = wglCreateContext(*hDC);             // create an OpenGL rendering context
    wglMakeCurrent(*hDC, hglrc);                // make it the current rendering context

    // Load OpenGL functions.
    LoadOpenGLFunctions();

    printf("OpenGL Version :%s\n", glGetString(GL_VERSION));
    printf("GLES Version   :%s\n", glGetString(GL_SHADING_LANGUAGE_VERSION));
    printf("GLU Version    :%s\n", glGetString(GLU_VERSION));
    printf("Vendor         :%s\n", glGetString(GL_VENDOR));
    printf("Renderer       :%s\n", glGetString(GL_RENDERER));

    //   +-----------------------------------------+
    //   |                  TEXTURE                |
    //   +-----------------------------------------+

    char source1[] = "#version 330\n"
        "precision mediump float;\n"
        "in vec3 v_vertex;\n"
        "in vec2 v_texture;\n"
        "out vec2 st;\n"
        "uniform mat4 m_matrix;\n"
        "void main()\n"
        "{\n"
        "st = v_texture;\n"
        "gl_Position = m_matrix * vec4(v_vertex, 1.0);\n"
        " }\n";

    char source2[] = "#version 330\n"
        "precision mediump float;\n"
        "in vec2 st;\n"
        "out vec4 FragColor;\n"
        "uniform sampler2D sampler;\n"
        "void main()\n"
        "{\n"
        "FragColor = texture(sampler, st);\n"
        "}\n";

    shader1.Create(source1, source2);
    square1.Create(shader1.Get(), (float)FRACTAL_WIDTH, (float)FRACTAL_HEIGHT);

    //   +-----------------------------------------+
    //   |          SELECTION RECTANGLE            |
    //   +-----------------------------------------+

    char source3[] = "#version 330\n"
        "precision mediump float;\n"
        "in vec3 v_vertex;\n"
        "uniform mat4 m_matrix;\n"
        "void main()\n"
        "{\n"
        "gl_Position = m_matrix * vec4(v_vertex, 1.0);\n"
        "}\n";

    char source4[] = "#version 330\n"
        "precision mediump float;\n"
        "out vec4 FragColor;\n"
        "void main()\n"
        "{\n"
        "FragColor = vec4(1.0, 1.0, 0.0, 1.0);\n"
        "}\n";

    shader2.Create(source3, source4);
    square2.Create(shader2.Get());

    //   +-----------------------------------------+
    //   |      PARAMETER INITIALIZATION           |
    //   +-----------------------------------------+

    hMenu = CreateAppMenu(hWnd);
    EnableMenuItem(hMenu, IDM_FRACTAL, MF_BYCOMMAND | MF_DISABLED);
    EnableMenuItem(hMenu, IDM_GO_BACK, MF_BYCOMMAND | MF_DISABLED);

    show_selection = is_dragging = false;

    Param1 = new double[PARAM_1_COUNT];
    Param2 = new int[PARAM_2_COUNT];
    Param3 = new unsigned char[PARAM_3_COUNT];

    Param2[0] = FRACTAL_WIDTH;
    Param2[1] = FRACTAL_HEIGHT;
    Param2[2] = MAX_ITERATION_COUNT;

    Param1[0] = 4.5;                                                    // cx
    Param1[1] = Param1[0] * ((double)Param2[1] / (double)Param2[0]);    // cy
    Param1[2] = -Param1[0] * 0.6;                                       // ox
    Param1[3] = -Param1[1] * 0.5;                                       // oy

    CreateColorGradient(Param3, MAX_ITERATION_COUNT);

    glGenTextures(1, &textures);
    glBindTexture(GL_TEXTURE_2D, textures);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, FRACTAL_WIDTH, FRACTAL_HEIGHT, 0, GL_RGBA, GL_UNSIGNED_BYTE, NULL);
    glGenerateMipmap(GL_TEXTURE_2D);

    glClearColor(0.0f, 0.0f, 0.0f, 0.0f);

    // do parallel computing after window is displayed
    PostMessage(hWnd, WM_AFTERWINDOWDISPLAYED, 0, 0);
}

void OnDestroy(HWND hWnd, HDC hDC)
{
    // release these objects.
    square1.Destroy();
    square2.Destroy();

    shader1.Destroy();
    shader2.Destroy();

    delete[] Param1;
    delete[] Param2;
    delete[] Param3;

    glDeleteTextures(1, &textures);

    DestroyAppMenu(hMenu);

    HGLRC hglrc;

    wglMakeCurrent(hDC, NULL);      // get current OpenGL rendering context
    hglrc = wglGetCurrentContext(); // make the rendering context not current
    wglDeleteContext(hglrc);        // delete the rendering context
    ReleaseDC(hWnd, hDC);           // releases a device context

    // close the program.
    PostQuitMessage(0);
}

void OnFileOpen(HWND hWnd)
{
    HRESULT hr;
    IFileOpenDialog* pFileOpen;
    FILEOPENDIALOGOPTIONS options;
    COMDLG_FILTERSPEC fs[2];
    IShellItem* pItem;
    PWSTR pszFile;
    wchar_t filename[MAX_PATH];
    wchar_t name[2][20], spec[2][6];
    char str[MAX_PATH];
    bool cancel;

    cancel = true;

    // Initialize COM.
    hr = CoInitializeEx(NULL, COINIT_APARTMENTTHREADED | COINIT_DISABLE_OLE1DDE);

    if (SUCCEEDED(hr)) {

        // Create the FileOpenDialog object.
        hr = CoCreateInstance(CLSID_FileOpenDialog, NULL, CLSCTX_ALL, IID_IFileOpenDialog, reinterpret_cast<void**>(&pFileOpen));

        if (SUCCEEDED(hr)) {

            // Filter.
            wcscpy_s(name[0], 20, L"Mandelbrot");
            wcscpy_s(spec[0], 6, L"*.mdl");

            fs[0].pszName = name[0];
            fs[0].pszSpec = spec[0];

            wcscpy_s(name[1], 20, L"All Files");
            wcscpy_s(spec[1], 6, L"*.*");

            fs[1].pszName = name[1];
            fs[1].pszSpec = spec[1];

            pFileOpen->SetFileTypes(2, fs);
            hr = pFileOpen->GetOptions(&options);

            if (SUCCEEDED(hr)) {
                options |= FOS_STRICTFILETYPES;
                pFileOpen->SetOptions(options);
            }

            // Show the Save dialog box.
            hr = pFileOpen->Show(hWnd);

            if (SUCCEEDED(hr)) {

                // Get the result object.
                hr = pFileOpen->GetResult(&pItem);

                if (SUCCEEDED(hr)) {

                    // Gets the filename that the user made in the dialog.
                    hr = pItem->GetDisplayName(SIGDN_FILESYSPATH, &pszFile);

                    // Copy the file name.
                    if (SUCCEEDED(hr)) {

                        cancel = false;

                        wcscpy_s(filename, MAX_PATH, pszFile);

                        // Release memory.
                        CoTaskMemFree(pszFile);
                    }

                    // Release result object.
                    pItem->Release();
                }
            }

            // Release FileOpenDialog object.
            pFileOpen->Release();
        }

        // Release COM.
        CoUninitialize();
    }

    if (cancel) return;

    if (!stack.Open(filename)) return;

    sprintf_s(str, MAX_PATH, "%s - %ws", szTitle, filename);
    SetWindowText(hWnd, str);

    EnableMenuItem(hMenu, IDM_GO_BACK, MF_BYCOMMAND | MF_ENABLED);
    OnViewGoBack(hWnd);
}

void OnFileSaveAs(HWND hWnd)
{
    HRESULT hr;
    IFileSaveDialog* pFileSave;
    FILEOPENDIALOGOPTIONS options;
    COMDLG_FILTERSPEC fs[2];
    IShellItem* pItem;
    PWSTR pszFile;
    UINT filetype;
    wchar_t filename[MAX_PATH];
    wchar_t name[2][20], spec[2][6];
    char str[MAX_PATH];
    bool cancel;

    cancel = true;

    // Initialize COM.
    hr = CoInitializeEx(NULL, COINIT_APARTMENTTHREADED | COINIT_DISABLE_OLE1DDE);

    if (SUCCEEDED(hr)) {

        // Create the FileSaveDialog object.
        hr = CoCreateInstance(CLSID_FileSaveDialog, NULL, CLSCTX_ALL, IID_IFileSaveDialog, reinterpret_cast<void**>(&pFileSave));

        if (SUCCEEDED(hr)) {

            // Set default filename extension
            pFileSave->SetDefaultExtension(L"mdl");

            // Set default filename.
            GetUniqueName(filename, MAX_PATH);
            pFileSave->SetFileName(filename);

            // Filter
            // 0
            wcscpy_s(name[0], 20, L"Mandelbrot");
            wcscpy_s(spec[0], 6, L"*.mdl");

            fs[0].pszName = name[0];
            fs[0].pszSpec = spec[0];

            // 1
            wcscpy_s(name[1], 20, L"All Files");
            wcscpy_s(spec[1], 6, L"*.*");

            fs[1].pszName = name[1];
            fs[1].pszSpec = spec[1];

            pFileSave->SetFileTypes(2, fs);
            hr = pFileSave->GetOptions(&options);

            if (SUCCEEDED(hr)) {
                options |= FOS_STRICTFILETYPES;
                pFileSave->SetOptions(options);
            }

            // Show the Save dialog box.
            hr = pFileSave->Show(hWnd);

            if (SUCCEEDED(hr)) {

                // Get the result object.
                hr = pFileSave->GetResult(&pItem);

                if (SUCCEEDED(hr)) {

                    // Gets the filename that the user made in the dialog.
                    hr = pItem->GetDisplayName(SIGDN_FILESYSPATH, &pszFile);

                    // Copy the file name.
                    if (SUCCEEDED(hr)) {

                        cancel = false;

                        wcscpy_s(filename, MAX_PATH, pszFile);

                        pFileSave->GetFileTypeIndex(&filetype);

                        // Release memory.
                        CoTaskMemFree(pszFile);
                    }

                    // Release result object.
                    pItem->Release();
                }
            }

            // Release FileSaveDialog object.
            pFileSave->Release();
        }

        // Release COM.
        CoUninitialize();
    }

    if (cancel) return;

    if (!stack.Save(filename, Param1[0], Param1[1], Param1[2], Param1[3])) return;

    sprintf_s(str, MAX_PATH, "%s - %ws", szTitle, filename);
    SetWindowText(hWnd, str);
}

void OnFileExit(HWND hWnd)
{
    DestroyWindow(hWnd);
}

void OnViewFractal(HWND hWnd)
{
    double x1, y1, x2, y2, sx, sy, cx, cy, ox, oy;

    EnableMenuItem(hMenu, IDM_FRACTAL, MF_BYCOMMAND | MF_DISABLED);
    EnableMenuItem(hMenu, IDM_GO_BACK, MF_BYCOMMAND | MF_ENABLED);

    show_selection = false;

    stack.Push(Param1[0], Param1[1], Param1[2], Param1[3]);

    x1 = ((double)px1 / (double)(Param2[0] - 1)) * Param1[0] + Param1[2];
    x2 = ((double)px2 / (double)(Param2[0] - 1)) * Param1[0] + Param1[2];

    y2 = ((double)py2 / (double)(Param2[1] - 1)) * Param1[1] + Param1[3];
    y1 = ((double)py1 / (double)(Param2[1] - 1)) * Param1[1] + Param1[3];

    sx = x2 - x1;
    sy = y2 - y1;

    cx = sx;
    cy = ((double)Param2[1] / (double)Param2[0]) * cx;

    if (cy < sy) {
        cy = sy;
        cx = ((double)Param2[0] / (double)Param2[1]) * cy;
    }

    ox = (x1 + x2 - cx) / 2.0;
    oy = (y1 + y2 - cy) / 2.0;

    Param1[0] = cx;
    Param1[1] = cy;
    Param1[2] = ox;
    Param1[3] = oy;

    DoParallelComputing(Param1, PARAM_1_SIZE, Param2, PARAM_2_SIZE, Param3, PARAM_3_SIZE);
}

void OnViewGoBack(HWND hWnd)
{
    stack.Pop(&Param1[0], &Param1[1], &Param1[2], &Param1[3]);

    show_selection = false;

    if (stack.IsEmpty())
        EnableMenuItem(hMenu, IDM_GO_BACK, MF_BYCOMMAND | MF_DISABLED);
    else
        EnableMenuItem(hMenu, IDM_GO_BACK, MF_BYCOMMAND | MF_ENABLED);

    DoParallelComputing(Param1, PARAM_1_SIZE, Param2, PARAM_2_SIZE, Param3, PARAM_3_SIZE);
}
